#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 15/05/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

namespace autd3 {
namespace gain {
namespace holo {

__device__ double absc2(const hipDoubleComplex x) { return x.x * x.x + x.y * x.y; }
__device__ double absc(const hipDoubleComplex x) { return sqrt(absc2(x)); }
__device__ hipDoubleComplex conj(const hipDoubleComplex a) { return make_hipDoubleComplex(a.x, -a.y); }
__device__ hipDoubleComplex mulc(const hipDoubleComplex a, const hipDoubleComplex b) {
  return make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__global__ void abs_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = make_hipDoubleComplex(absc(a[idx]), 0.0);
}
__global__ void abs_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, double* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = absc(a[idx]);
}

void cu_abs(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}
void cu_abs(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, double* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void sqrt_kernel(const double* a, const uint32_t row, const uint32_t col, double* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = sqrt(a[idx]);
}

void cu_sqrt(const double* a, const uint32_t row, const uint32_t col, double* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  sqrt_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void conj_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

void cu_conj(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void arg_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  const double s = absc(a[idx]);
  const double x = a[idx].x / s;
  const double y = a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_arg(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  arg_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void reciprocal_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  double s = absc2(a[idx]);
  const double x = a[idx].x / s;
  const double y = -a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_reciprocal(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

__device__ hipDoubleComplex expc(const hipDoubleComplex x) {
  const double s = exp(x.x);
  const double r = cos(x.y);
  const double i = sin(x.y);
  return make_hipDoubleComplex(s * r, s * i);
}

__global__ void exp_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = expc(a[idx]);
}

void cu_exp(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  exp_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void pow_kernel(const double* a, const double p, const uint32_t row, const uint32_t col, double* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = pow(a[idx], p);
}

void cu_pow(const double* a, const double p, const uint32_t row, const uint32_t col, double* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  pow_kernel<<<grid, block>>>(a, p, row, col, b);
}

__global__ void imag_kernel(const hipDoubleComplex* src, const uint32_t row, const uint32_t col, double* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].y;
}

void cu_imag(const hipDoubleComplex* src, const uint32_t row, const uint32_t col, double* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  imag_kernel<<<grid, block>>>(src, row, col, dst);
}

__global__ void real_kernel(const hipDoubleComplex* src, const uint32_t row, const uint32_t col, double* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = src[idx].x;
}

void cu_real(const hipDoubleComplex* src, const uint32_t row, const uint32_t col, double* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  real_kernel<<<grid, block>>>(src, row, col, dst);
}

__global__ void make_complex_kernel(const double* re, const double* im, const uint32_t row, const uint32_t col, hipDoubleComplex* dst) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  dst[idx] = make_hipDoubleComplex(re[idx], im[idx]);
}

void cu_make_complex(const double* re, const double* im, const uint32_t row, const uint32_t col, hipDoubleComplex* dst) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(re, im, row, col, dst);
}

__global__ void set_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : make_hipDoubleComplex(0.0, 0.0);
}

void cu_set_diagonal(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void get_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

void cu_get_diagonal(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void get_diagonal_kernel(const double* a, uint32_t row, uint32_t col, double* b) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    unsigned int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

void cu_get_diagonal(const double* a, const uint32_t row, const uint32_t col, double* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void hadamard_product_kernel(const hipDoubleComplex* a, const hipDoubleComplex* b, const uint32_t row, const uint32_t col,
                                        hipDoubleComplex* c) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  unsigned int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

void cu_hadamard_product(const hipDoubleComplex* a, const hipDoubleComplex* b, const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(0.0, 0.0);
}

void cu_calc_singular_inv(double* d_s, const uint32_t row, const uint32_t col, const double alpha, hipDoubleComplex* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t row, uint32_t col, double alpha, double* p_singular_inv) {
  unsigned int xi = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = d_s[xi] / (d_s[xi] * d_s[xi] + alpha);
  else
    p_singular_inv[yi + xi * row] = 0.0;
}

void cu_calc_singular_inv(double* d_s, const uint32_t row, const uint32_t col, const double alpha, double* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

__global__ void col_sum_kernel(const double* din, uint32_t m, uint32_t n, double* dout) {
  extern __shared__ double smem[];

  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  double local_sum = i < n ? din[i * m + row] : 0;
  if (i + blockDim.x < n) {
    local_sum += din[(i + blockDim.x) * m + row];
  }
  smem[tid] = local_sum;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) {
      smem[tid] = local_sum = local_sum + smem[tid + s];
    }
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) {
      local_sum += smem[tid + 32];
    }
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) {
      local_sum += __shfl_down_sync(0xffffffff, local_sum, offset);
    }
  }
  if (tid == 0) {
    dout[blockIdx.x * m + row] = local_sum;
  }
}

void cu_reduce_col(const double* mat, const uint32_t m, const uint32_t n, double* result, double* buffer) {
  dim3 block(BLOCK_SIZE / 2, 1, 1);
  dim3 grid((n - 1) / BLOCK_SIZE + 1, m, 1);
  col_sum_kernel<<<grid, block, BLOCK_SIZE * sizeof(double)>>>(mat, m, n, buffer);
  col_sum_kernel<<<dim3(1, m, 1), dim3(max((grid.x + 1) / 2, 1), 1, 1), max(grid.x, 2) * sizeof(double)>>>(buffer, m, grid.x, result);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
