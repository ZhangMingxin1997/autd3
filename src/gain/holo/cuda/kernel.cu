#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 14/05/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

namespace autd3 {
namespace gain {
namespace holo {

__device__ double absc2(hipDoubleComplex x) { return x.x * x.x + x.y * x.y; }
__device__ double absc(hipDoubleComplex x) { return sqrt(absc2(x)); }
__device__ hipDoubleComplex conj(hipDoubleComplex a) { return make_hipDoubleComplex(a.x, -a.y); }
__device__ hipDoubleComplex mulc(hipDoubleComplex a, hipDoubleComplex b) { return make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x); }

__global__ void abs_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = make_hipDoubleComplex(absc(a[idx]), 0.0);
}

void cu_abs(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void conj_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

void cu_conj(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void arg_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  double s = absc(a[idx]);
  double x = a[idx].x / s;
  double y = a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_arg(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  arg_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void reciprocal_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  double s = absc2(a[idx]);
  double x = a[idx].x / s;
  double y = -a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_reciprocal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void set_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : make_hipDoubleComplex(0.0, 0.0);
}

void cu_set_diagonal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void hadamard_product_kernel(const hipDoubleComplex* a, const hipDoubleComplex* b, const uint32_t row, const uint32_t col,
                                        hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

void cu_hadamard_product(const hipDoubleComplex* a, const hipDoubleComplex* b, uint32_t row, uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(0.0, 0.0);
}

void cu_calc_singular_inv(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

__global__ void col_sum_kernel(const hipDoubleComplex* din, uint32_t m, uint32_t n, hipDoubleComplex* dout) {
  extern __shared__ double smem[];

  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (row >= m) return;

  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  double local_sum_r = (i < n) ? din[i * m + row].x : 0;
  double local_sum_i = (i < n) ? din[i * m + row].y : 0;
  if (i + blockDim.x < n) {
    local_sum_r += din[(i + blockDim.x) * m + row].x;
    local_sum_i += din[(i + blockDim.x) * m + row].y;
  }
  smem[2 * tid] = local_sum_r;
  smem[2 * tid + 1] = local_sum_i;
  __syncthreads();

  for (unsigned int s = blockDim.x >> 1; s > 32; s >>= 1) {
    if (tid < s) {
      smem[2 * tid] = local_sum_r = local_sum_r + smem[2 * (tid + s)];
      smem[2 * tid + 1] = local_sum_i = local_sum_i + smem[2 * (tid + s) + 1];
    }
    __syncthreads();
  }
  if (tid < 32) {
    if (blockDim.x >= 64) {
      local_sum_r += smem[2 * (tid + 32)];
      local_sum_i += smem[2 * (tid + 32) + 1];
    }
    for (int offset = 32 >> 1; offset > 0; offset >>= 1) {
      local_sum_r += __shfl_down_sync(0xffffffff, local_sum_r, offset);
      local_sum_i += __shfl_down_sync(0xffffffff, local_sum_i, offset);
    }
  }
  if (tid == 0) {
    dout[blockIdx.x * m + row].x = local_sum_r;
    dout[blockIdx.x * m + row].y = local_sum_i;
  }
}

void cu_reduce_col(const hipDoubleComplex* mat, uint32_t m, uint32_t n, hipDoubleComplex* result, hipDoubleComplex* buffer) {
  dim3 block(BLOCK_SIZE / 2, 1, 1);
  dim3 grid((n - 1) / BLOCK_SIZE + 1, m, 1);

  col_sum_kernel<<<grid, block, BLOCK_SIZE * sizeof(hipDoubleComplex)>>>(mat, m, n, buffer);
  col_sum_kernel<<<dim3(1, m, 1), dim3(max((grid.x + 1) / 2, 1), 1, 1), max(grid.x, 2) * sizeof(hipDoubleComplex)>>>(buffer, m, grid.x, result);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
