#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 14/05/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

namespace autd3 {
namespace gain {
namespace holo {

__device__ hipDoubleComplex conj(hipDoubleComplex a) { return make_hipDoubleComplex(a.x, -a.y); }

__global__ void conj_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

void cu_conj(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void set_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : make_hipDoubleComplex(0.0, 0.0);
}

void cu_set_diagonal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(0.0, 0.0);
}

void cu_calc_singular_inv(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
