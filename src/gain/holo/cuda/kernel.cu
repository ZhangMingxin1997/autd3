#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 13/05/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

namespace autd3 {
namespace gain {
namespace holo {

__global__ void make_complex_kernel(const double* r, const double* i, const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= row || yi >= col) return;

  int idx = xi + yi * row;
  c[idx] = make_hipDoubleComplex(r[idx], i[idx]);
}

void cu_make_complex(const double* r, const double* i, const uint32_t row, const uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((row - 1) / BLOCK_SIZE + 1, (col - 1) / BLOCK_SIZE + 1, 1);
  make_complex_kernel<<<grid, block>>>(r, i, row, col, c);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
