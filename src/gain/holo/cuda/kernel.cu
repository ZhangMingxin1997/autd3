#include "hip/hip_runtime.h"
/*
 * File: kernel.cu
 * Project: cuda
 * Created Date: 13/05/2022
 * Author: Shun Suzuki
 * -----
 * Last Modified: 14/05/2022
 * Modified By: Shun Suzuki (suzuki@hapis.k.u-tokyo.ac.jp)
 * -----
 * Copyright (c) 2022 Hapis Lab. All rights reserved.
 *
 */

#include <hip/hip_runtime_api.h>

#include <complex>

#include "./kernel.h"

namespace autd3 {
namespace gain {
namespace holo {

__device__ double absc2(hipDoubleComplex x) { return x.x * x.x + x.y * x.y; }
__device__ double absc(hipDoubleComplex x) { return sqrt(absc2(x)); }
__device__ hipDoubleComplex conj(hipDoubleComplex a) { return make_hipDoubleComplex(a.x, -a.y); }
__device__ hipDoubleComplex mulc(hipDoubleComplex a, hipDoubleComplex b) { return make_hipDoubleComplex(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x); }

__global__ void abs_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = make_hipDoubleComplex(absc(a[idx]), 0.0);
}

void cu_abs(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  abs_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void conj_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = conj(a[idx]);
}

void cu_conj(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  conj_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void arg_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  double s = absc(a[idx]);
  double x = a[idx].x / s;
  double y = a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_arg(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  arg_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void reciprocal_kernel(const hipDoubleComplex* a, const uint32_t row, const uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  double s = absc2(a[idx]);
  double x = a[idx].x / s;
  double y = -a[idx].y / s;
  b[idx] = make_hipDoubleComplex(x, y);
}

void cu_reciprocal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  reciprocal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void set_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  b[idx] = xi == yi ? a[xi] : make_hipDoubleComplex(0.0, 0.0);
}

void cu_set_diagonal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  set_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void get_diagonal_kernel(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi) {
    int idx = yi + xi * row;
    b[xi] = a[idx];
  }
}

void cu_get_diagonal(const hipDoubleComplex* a, uint32_t row, uint32_t col, hipDoubleComplex* b) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  get_diagonal_kernel<<<grid, block>>>(a, row, col, b);
}

__global__ void hadamard_product_kernel(const hipDoubleComplex* a, const hipDoubleComplex* b, const uint32_t row, const uint32_t col,
                                        hipDoubleComplex* c) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  int idx = yi + xi * row;
  c[idx] = mulc(a[idx], b[idx]);
}

void cu_hadamard_product(const hipDoubleComplex* a, const hipDoubleComplex* b, uint32_t row, uint32_t col, hipDoubleComplex* c) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  hadamard_product_kernel<<<grid, block>>>(a, b, row, col, c);
}

__global__ void calc_singular_inv_kernel(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  int xi = blockIdx.x * blockDim.x + threadIdx.x;
  int yi = blockIdx.y * blockDim.y + threadIdx.y;
  if (xi >= col || yi >= row) return;

  if (xi == yi)
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(d_s[xi] / (d_s[xi] * d_s[xi] + alpha), 0.0);
  else
    p_singular_inv[yi + xi * row] = make_hipDoubleComplex(0.0, 0.0);
}

void cu_calc_singular_inv(double* d_s, uint32_t row, uint32_t col, double alpha, hipDoubleComplex* p_singular_inv) {
  dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 grid((col - 1) / BLOCK_SIZE + 1, (row - 1) / BLOCK_SIZE + 1, 1);
  calc_singular_inv_kernel<<<grid, block>>>(d_s, row, col, alpha, p_singular_inv);
}

}  // namespace holo
}  // namespace gain
}  // namespace autd3
